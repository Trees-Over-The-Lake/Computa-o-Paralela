#include "hip/hip_runtime.h"

/*
Tempo sequencial:

real    0m31,569s
user    0m31,408s
sys     0m0,077s

real    0m31,120s
user    0m30,967s
sys     0m0,087s

Tempo Paralelo - Multicore:

real    0m9,857s
user    1m1,442s
sys     0m2,972s

real    0m9,663s
user    1m1,498s
sys     0m2,830s

real    0m9,709s
user    1m1,580s
sys     0m3,002s

Tempo paralelo - GPU

real    0m31,675s
user    0m31,759s
sys     0m0,050s

real    0m31,450s
user    0m31,600s
sys     0m0,043s

For paralelo

real    0m9,370s
user    0m54,341s
sys     0m0,196s

real    0m9,421s
user    0m55,248s
sys     0m0,227s

For paralelo com SIMD ativado

real    0m9,144s
user    0m56,485s
sys     0m0,878s

real    0m8,984s
user    0m56,533s
sys     0m0,232s

CUDA:

real    0m0,752s
user    0m0,588s
sys     0m0,138s

real    0m0,693s
user    0m0,559s
sys     0m0,114s

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void mm(double* a, double* b, double* c, int width) {
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int col = blockIdx.x*blockDim.x+threadIdx.x;

	if((row < width) && (col < width)) {	
		double sum = 0;
		for (int k = 0; k < width; k++) {
			double x = a[i * width + k];
			double y = b[k * width + j];
			sum += x * y;
		}
		c[i * width + j] = sum;
	}
}

int main() {
	int width = 2000;
	double *a = (double*) malloc (width * width * sizeof(double));
	double *b = (double*) malloc (width * width * sizeof(double));
	double *c = (double*) malloc (width * width * sizeof(double));

	for(int i = 0; i < width; i++) {	  
		for(int j = 0; j < width; j++) {
			a[i*width+j] = i;
			b[i*width+j] = j;
			c[i*width+j] = 0;
		}
	}

	int size = width*width*sizeof(double);
	double *d_a, *d_b, *d_c;

	hipMalloc((void **) &d_a, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_b, size);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_c, size);

	int block_size = 32;
	dim3 dimGrid((width-1)/block_size+1, (width-1)/block_size+1, 1);
	dim3 dimBlock(block_size, block_size, 1);

	mm<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	/*
	for(int i = 0; i < width; i++) {
		for(int j = 0; j < width; j++) {
			printf("\n c[%d][%d] = %lf",i,j,c[i*width+j]);
		}
	}
	*/

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}