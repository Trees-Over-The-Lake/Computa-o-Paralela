#include "hip/hip_runtime.h"
#include <algorithm>
#include "../headers/block.cu"
#include "../headers/sha256.cuh"
#include "../headers/sha256Host.cuh"

#define THREADS 1500
#define BLOCKS 256
#define DIFFICULTY 3
#define RANDOM_LEN 20

typedef unsigned char BYTE;

__constant__ BYTE characterSet[63] = {"abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ1234567890"};

__device__ unsigned long long deviceRandomGen(unsigned long long x) {
    x ^= (x << 21);
    x ^= (x >> 35);
    x ^= (x << 4);
    return x;
}

__global__ void sha256_cuda(BYTE* input, BYTE* solution, int* blockContainsSolution, unsigned long long baseSeed, size_t textLen) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long seed = baseSeed;
    seed += (unsigned long long) i;

    BYTE digest[32];
    BYTE random[RANDOM_LEN];
    memset(digest, 0, SHA256::DIGEST_SIZE);
    memset(random, 0, RANDOM_LEN);

    for(int j = 0; j < RANDOM_LEN; j++) {
        seed = deviceRandomGen(seed);
        int randomIdx = (int) (seed % 62);
        random[j] = characterSet[randomIdx];
    }

    SHA256CUDA::sha256(input, random, textLen, RANDOM_LEN, digest);

    for(int j = 0; j < DIFFICULTY; j++) 
        if(digest[j] > 0)
            return 0;
    
    if(*blockContainsSolution == 1) 
        return;

    *blockContainsSolution = 1;

    for(int j = 0; j < RANDOM_LEN; j++) 
        solution[j] = random[j];
}

__global__ void initSolutionMemory() {