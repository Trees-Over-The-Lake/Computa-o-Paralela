#include "hip/hip_runtime.h"
#include ""
#include "headers/blockchain.cuh"

#include <iostream>

int main() {
	uint32_t nDifficulty = 5;
	Blockchain bChain = Blockchain(nDifficulty);

	std::cout << "Mining block 1..." << std::endl;
	bChain.AddBlock(new Block(1, "Block 1 Data"));

	std::cout << "Mining block 2..." << std::endl;
	bChain.AddBlock(new Block(2, "Block 2 Data"));

	std::cout << "Mining block 3..." << std::endl;
	bChain.AddBlock(new Block(3, "Block 3 Data"));

	bChain.PrintBlockchain();
	return 0;
}