
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
******************************
* Tempo do código sequencial *
******************************

real    0m0.437s
user    0m0.197s
sys     0m0.229s

real    0m0.393s
user    0m0.184s
sys     0m0.207s

real    0m0.393s
user    0m0.180s
sys     0m0.211s
*/

/*
*****************************
* Tempo paralelo GPU - CUDA *
*****************************

real    0m1.258s
user    0m0.119s
sys     0m1.053s

real    0m1.112s
user    0m0.117s
sys     0m0.896s

real    0m1.220s
user    0m0.130s
sys     0m0.999s
*/

__global__ void scan_cuda(double* a, double *s, int width) {
  // kernel scan
  int t = threadIdx.x;
  int b = blockIdx.x * blockDim.x;

  double x;

  __shared__ double p[1024];

  if(b+t < width)
    p[t] = a[b+t];

  __syncthreads();

  for(int i = 1; i < blockDim.x; i *= 2) {
    if (t >= i)
      x = p[t] + p[t-i];

      __syncthreads();

      if (t >= i)
        p[t] = x;

      __syncthreads();
  }

  if (b + t < width)
    a[b+t] = p[t];

  if(t == blockDim.x - 1)
    s[blockIdx.x + 1] = a[b+t];
}

__global__ void add_cuda(double *a, double *s, int width) {
  // kernel add
  int t = threadIdx.x;
  int b = blockIdx.x * blockDim.x;

  if (b+t < width)
    a[b+t] += s[blockIdx.x];
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));

  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocar vetores "a" e "s" no device
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_s, size);

  // copiar vetor "a" para o device
  hipMemcpy(&d_a, a, size, hipMemcpyHostToDevice);

  // definição do número de blocos e threads (dimGrid e dimBlock)
  dim3 dimGrid(num_blocks, 1, 1);
  dim3 dimBlock(block_size, 1, 1);

  // chamada do kernel scan
  scan_cuda<<<dimGrid, dimBlock>>>(d_a, d_s, width);

  // copiar vetor "s" para o host
  hipMemcpy(&d_s, s, size, hipMemcpyDeviceToHost);

  // scan no host (já implementado)
  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];

  // copiar vetor "s" para o device
  hipMemcpy(&d_s, s, size, hipMemcpyHostToDevice);

  // chamada do kernel da soma
  add_cuda<<<dimGrid, dimBlock>>>(a, s, width);

  // copiar o vetor "a" para o host
  hipMemcpy(&d_a, a, size, hipMemcpyDeviceToHost);

  printf("\na[%d] = %f\n",width-1,a[width-1]);

  hipFree(d_a);
  hipFree(d_s);
}